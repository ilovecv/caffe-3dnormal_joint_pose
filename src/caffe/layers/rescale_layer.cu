#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {


template <typename Dtype>
Dtype RescaleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
    
    Forward_cpu(bottom, top);
  
  return Dtype(0);
}

// TODO(Yangqing): implement the GPU version of softmax.
template <typename Dtype>
void RescaleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
    
    Backward_cpu(top, propagate_down, bottom);
    
}

INSTANTIATE_CLASS(RescaleLayer);

}  // namespace caffe
