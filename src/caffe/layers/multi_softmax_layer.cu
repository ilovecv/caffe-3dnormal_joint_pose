#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;

namespace caffe {

template <typename Dtype>
__global__ void kernel_get_max(const int num, const int dim,
    const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num) {
    Dtype maxval = -FLT_MAX;
    for (int i = 0; i < dim; ++i) {
      maxval = max(data[index * dim + i], maxval);
    }
    out[index] = maxval;
  }
}

template <typename Dtype>
__global__ void kernel_softmax_div(const int num, const int dim,
    const Dtype* scale, Dtype* data) {
  CUDA_KERNEL_LOOP(index, num * dim) {
    int n = index / dim;
    data[index] /= scale[n];
  }
}

template <typename Dtype>
__global__ void kernel_exp(const int num, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, num) {
    out[index] = exp(data[index]);
  }
}

template <typename Dtype>
Dtype MultiSoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
    return Forward_cpu(bottom,top);
}

// TODO(Yangqing): implement the GPU version of softmax.
template <typename Dtype>
void MultiSoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
    
}

INSTANTIATE_CLASS(MultiSoftmaxLayer);


}  // namespace caffe
