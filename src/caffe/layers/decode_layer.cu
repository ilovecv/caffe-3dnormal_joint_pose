// Copyright 2014 BVLC and contributors.

#include <hipblas.h>

#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
Dtype DecodeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  Forward_cpu(bottom, top);
  return Dtype(0);
}

template <typename Dtype>
void DecodeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
    Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_CLASS(DecodeLayer);

}  // namespace caffe
