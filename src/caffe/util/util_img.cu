#include "hip/hip_runtime.h"

/**
 * developed by zhujin
 */
#include <google/protobuf/text_format.h>
#include <google/protobuf/io/zero_copy_stream_impl.h>
#include <google/protobuf/io/coded_stream.h>


#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/util_img.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/proto/caffe.pb.h"



namespace caffe {

template <typename Dtype>
__global__ void kernel_BiLinearResize(const int nthreads, const Dtype* src_data, const int src_height, const int src_width,
		Dtype* dst_data, const int dst_height, const int dst_width, const Dtype scale_h, const Dtype scale_w)
{

	CUDA_KERNEL_LOOP(i, nthreads) {
		int dst_h = i /dst_width;
		Dtype fh = dst_h * scale_h;
		const int src_h = floor(fh);
		fh -= src_h;
		const Dtype w_h0 = std::abs(1.0f - fh);
		const Dtype w_h1 = std::abs(fh);

		const int dst_offset_1 =  dst_h * dst_width;
		const int src_offset_1 =  src_h * src_width;

		int dst_w = i %dst_width;
		Dtype fw = dst_w * scale_w;
		const int src_w = floor(fw);
		fw -= src_w;
		const Dtype w_w0 = std::abs(1.0f - fw);
		const Dtype w_w1 = std::abs(fw);

		const int dst_idx = dst_offset_1 + dst_w;
		dst_data[dst_idx] = 0;

		const int src_idx = src_offset_1 + src_w;

		dst_data[dst_idx] += (w_h0 * w_w0 * src_data[src_idx]);
		if (src_w + 1 < src_width)
			dst_data[dst_idx] += (w_h0 * w_w1 * src_data[src_idx + 1]);
		if (src_h + 1 < src_height)
			dst_data[dst_idx] += (w_h1 * w_w0 * src_data[src_idx + src_width]);

		if (src_w + 1 < src_width && src_h + 1 < src_height)
			dst_data[dst_idx] += (w_h1 * w_w1 * src_data[src_idx + src_width + 1]);
	}
}


template <typename Dtype>
void BiLinearResizeMat_gpu(const Dtype* src, const int src_height, const int src_width,
		Dtype* dst, const int dst_height, const int dst_width)
{
	const Dtype scale_w = src_width / (Dtype)dst_width;
	const Dtype scale_h = src_height / (Dtype)dst_height;


	int loop_n = dst_height * dst_width;
	kernel_BiLinearResize<Dtype> <<<CAFFE_GET_BLOCKS(loop_n), CAFFE_CUDA_NUM_THREADS >>>(
			loop_n,src, src_height, src_width, dst, dst_height, dst_width, scale_h, scale_w);

	//CUDA_POST_KERNEL_CHECK;
}


template void BiLinearResizeMat_gpu(const float* src, const int src_height, const int src_width,
		float* dst, const int dst_height, const int dst_width);

template void BiLinearResizeMat_gpu(const double* src, const int src_height, const int src_width,
		double* dst, const int dst_height, const int dst_width);



template <typename Dtype>
void ResizeBlob_gpu(const Blob<Dtype>* src, const int src_n, const int src_c,
		Blob<Dtype>* dst, const int dst_n, const int dst_c) {


	const int src_channels = src->channels();
	const int src_height = src->height();
	const int src_width = src->width();
	const int src_offset = (src_n * src_channels + src_c) * src_height * src_width;

	const int dst_channels = dst->channels();
	const int dst_height = dst->height();
	const int dst_width = dst->width();
	const int dst_offset = (dst_n * dst_channels + dst_c) * dst_height * dst_width;

	const Dtype* src_data = &(src->gpu_data()[src_offset]);
	Dtype* dst_data = &(dst->mutable_gpu_data()[dst_offset]);
	BiLinearResizeMat_gpu(src_data,  src_height,  src_width,
			dst_data,  dst_height,  dst_width);
	CUDA_POST_KERNEL_CHECK;
}

template void ResizeBlob_gpu(const Blob<float>* src, const int src_n, const int src_c,
		Blob<float>* dst, const int dst_n, const int dst_c);
template void ResizeBlob_gpu(const Blob<double>* src, const int src_n, const int src_c,
		Blob<double>* dst, const int dst_n, const int dst_c);

template <typename Dtype>
__global__ void kernel_GetBiLinearResizeMatRules(const int nthreads,  const int src_height, const int src_width,
		const int dst_height, const int dst_width, const Dtype scale_h, const Dtype scale_w,
		Dtype* loc1, Dtype* weight1, Dtype* loc2, Dtype* weight2,
				Dtype* loc3, Dtype* weight3, Dtype* loc4, Dtype* weight4)
{
	CUDA_KERNEL_LOOP(index, nthreads)
	{
		int dst_h = index /dst_width;
		Dtype fh = dst_h * scale_h;
		const int src_h = floor(fh);
		fh -= src_h;
		const Dtype w_h0 = std::abs(1.0f - fh);
		const Dtype w_h1 = std::abs(fh);

		const int dst_offset_1 =  dst_h * dst_width;
		const int src_offset_1 =  src_h * src_width;

		int dst_w = index %dst_width;
		Dtype fw = dst_w * scale_w;
		const int src_w = floor(fw);
		fw -= src_w;
		const Dtype w_w0 = std::abs(1.0f - fw);
		const Dtype w_w1 = std::abs(fw);

		const int dst_idx = dst_offset_1 + dst_w;
//		dst_data[dst_idx] = 0;

		const int src_idx = src_offset_1 + src_w;

		loc1[dst_idx] = src_idx;
		weight1[dst_idx] = w_h0 * w_w0;

		if (src_w + 1 < src_width)
		{
			loc2[dst_idx] = src_idx + 1;
			weight2[dst_idx] = w_h0 * w_w1;
//			dst_data[dst_idx] += (w_h0 * w_w1 * src_data[src_idx + 1]);
		}

		if (src_h + 1 < src_height)
		{
//			dst_data[dst_idx] += (w_h1 * w_w0 * src_data[src_idx + src_width]);
			weight3[dst_idx] = w_h1 * w_w0;
			loc3[dst_idx] = src_idx + src_width;
		}

		if (src_w + 1 < src_width && src_h + 1 < src_height)
		{
			loc4[dst_idx] = src_idx + src_width + 1;
			weight4[dst_idx] = w_h1 * w_w1;
//			dst_data[dst_idx] += (w_h1 * w_w1 * src_data[src_idx + src_width + 1]);
		}

	}
}



template <typename Dtype>
__global__ void kernel_ResizeBlob(const int nthreads,const int num,const int channels, const Dtype* src, const int src_height, const int src_width,
		Dtype* dst, const int dst_height, const int dst_width, const Dtype scale_h, const Dtype scale_w)
{
	CUDA_KERNEL_LOOP(index, nthreads) {
		int i = index %( dst_height * dst_width);
		int c = (index/(dst_height * dst_width))%channels;
		int n = (index/(dst_height * dst_width))/channels;
		int src_offset = (n * channels + c) * src_height * src_width;
		int dst_offset = (n * channels + c) * dst_height * dst_width;

		const Dtype* src_data = src+src_offset;
		Dtype* dst_data = dst+dst_offset;

		int dst_h = i /dst_width;
		Dtype fh = dst_h * scale_h;
		const int src_h = floor(fh);
		fh -= src_h;
		const Dtype w_h0 = std::abs(1.0f - fh);
		const Dtype w_h1 = std::abs(fh);

		const int dst_offset_1 =  dst_h * dst_width;
		const int src_offset_1 =  src_h * src_width;

		int dst_w = i %dst_width;
		Dtype fw = dst_w * scale_w;
		const int src_w = floor(fw);
		fw -= src_w;
		const Dtype w_w0 = std::abs(1.0f - fw);
		const Dtype w_w1 = std::abs(fw);

		const int dst_idx = dst_offset_1 + dst_w;
		dst_data[dst_idx] = 0;

		const int src_idx = src_offset_1 + src_w;

		dst_data[dst_idx] += (w_h0 * w_w0 * src_data[src_idx]);
		if (src_w + 1 < src_width)
			dst_data[dst_idx] += (w_h0 * w_w1 * src_data[src_idx + 1]);
		if (src_h + 1 < src_height)
			dst_data[dst_idx] += (w_h1 * w_w0 * src_data[src_idx + src_width]);

		if (src_w + 1 < src_width && src_h + 1 < src_height)
			dst_data[dst_idx] += (w_h1 * w_w1 * src_data[src_idx + src_width + 1]);

	}
}


template <typename Dtype>
void ResizeBlob_gpu(const Blob<Dtype>* src,Blob<Dtype>* dst) {

	CHECK(src->num() == dst->num())<<"src->num() == dst->num()";
	CHECK(src->channels() == dst->channels())<< "src->channels() == dst->channels()";

	const int src_num = src->num();
	const int src_channels = src->channels();
	const int src_height = src->height();
	const int src_width = src->width();


	const int dst_channels = dst->channels();
	const int dst_height = dst->height();
	const int dst_width = dst->width();


	const Dtype scale_w = src_width / (Dtype)dst_width;
	const Dtype scale_h = src_height / (Dtype)dst_height;
	int loop_n = dst_height * dst_width*dst_channels*src_num;
	const Dtype* src_data = src->gpu_data();
	Dtype* dst_data = dst->mutable_gpu_data();
	kernel_ResizeBlob<Dtype> <<<CAFFE_GET_BLOCKS(loop_n), CAFFE_CUDA_NUM_THREADS >>>(loop_n,src_num,src_channels,
			src_data, src_height,src_width,
			dst_data, dst_height, dst_width,
			scale_h,scale_w);
	CUDA_POST_KERNEL_CHECK;
}



template void ResizeBlob_gpu(const Blob<float>* src,
		Blob<float>* dst);
template void ResizeBlob_gpu(const Blob<double>* src,
		Blob<double>* dst);


template <typename Dtype>
void GetBiLinearResizeMatRules_gpu( const int src_height, const int src_width,
		 const int dst_height, const int dst_width,
		Dtype* loc1, Dtype* weight1, Dtype* loc2, Dtype* weight2,
		Dtype* loc3, Dtype* weight3, Dtype* loc4, Dtype* weight4)
{
	const Dtype scale_w = src_width / (Dtype)dst_width;
	const Dtype scale_h = src_height / (Dtype)dst_height;


	int loop_n = dst_height * dst_width;
	caffe::caffe_gpu_set(loop_n,(Dtype)0,loc1);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,loc2);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,loc4);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,loc3);

	caffe::caffe_gpu_set(loop_n,(Dtype)0,weight1);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,weight2);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,weight3);
	caffe::caffe_gpu_set(loop_n,(Dtype)0,weight4);
	kernel_GetBiLinearResizeMatRules<Dtype> <<<CAFFE_GET_BLOCKS(loop_n), CAFFE_CUDA_NUM_THREADS >>>(
			loop_n,  src_height,  src_width,
			dst_height, dst_width, scale_h, scale_w,
			loc1,  weight1,  loc2,  weight2,
			loc3,  weight3,   loc4,   weight4);
	CUDA_POST_KERNEL_CHECK;
}

template void GetBiLinearResizeMatRules_gpu(  const int src_height, const int src_width,
		 const int dst_height, const int dst_width,
		float* loc1, float* weight1, float* loc2, float* weight2,
				float* loc3, float* weight3, float* loc4, float* weight4);

template void GetBiLinearResizeMatRules_gpu(  const int src_height, const int src_width,
		 const int dst_height, const int dst_width,
		double* loc1, double* weight1, double* loc2, double* weight2,
				double* loc3, double* weight3, double* loc4, double* weight4);



template <typename Dtype>
void ResizeBlob_gpu(const Blob<Dtype>* src,Blob<Dtype>* dst,
		Blob<Dtype>* loc1, Blob<Dtype>* loc2, Blob<Dtype>* loc3, Blob<Dtype>* loc4){

	CHECK(src->num() == dst->num())<<"src->num() == dst->num()";
	CHECK(src->channels() == dst->channels())<< "src->channels() == dst->channels()";

	GetBiLinearResizeMatRules_gpu(  src->height(),src->width(),
			 dst->height(), dst->width(),
			loc1->mutable_gpu_data(), loc1->mutable_gpu_diff(), loc2->mutable_gpu_data(), loc2->mutable_gpu_diff(),
			loc3->mutable_gpu_data(), loc3->mutable_gpu_diff(), loc4->mutable_gpu_data(), loc4->mutable_gpu_diff());

	for(int n=0;n< src->num();++n)
	{
		for(int c=0; c < src->channels() ; ++c)
		{
			ResizeBlob_gpu(src,n,c,dst,n,c);
		}
	}
}
template void ResizeBlob_gpu(const Blob<float>* src,Blob<float>* dst,
		Blob<float>* loc1, Blob<float>* loc2, Blob<float>* loc3, Blob<float>* loc4);
template void ResizeBlob_gpu(const Blob<double>* src,Blob<double>* dst,
		Blob<double>* loc1, Blob<double>* loc2, Blob<double>* loc3, Blob<double>* loc4);


// namespace caffe
}
